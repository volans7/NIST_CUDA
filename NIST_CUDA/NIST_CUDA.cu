#include "hip/hip_runtime.h"
﻿#include <iostream> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 

#include <math.h>
#include <string.h>
#include "include\decls.h"
#include "include\cephes.h"  
#include "include\utilities.h"

#pragma comment(lib, "cudart") 

void	partitionResultFile(int numOfFiles, int numOfSequences, int option, int testNameID);
void	postProcessResults(int option);
int		cmp(const double *a, const double *b);
int		computeMetrics(char *s, int test);
 
using namespace std; 

#define N 10


__global__ void add2( int *a, int *b, int *c)
{
	int tid = blockIdx.x; // это CPU 0
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

__global__ void add(int a, int b, int *c) // ключевое слово, означающее, что функция должа испольняться на GPU
{ 
    *c=a + b;
} 

int DeviceInfo(void)
{
	int count;
	hipDeviceProp_t prop;

	//Информация об устройстве
	hipGetDeviceCount( &count );
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties (&prop, i);
		printf("--- Общая информация об устройстве %d ---\n", i);
		printf("Имя: %s\n", prop.name);
		printf("Вычислительные возможности: %d.%d\n", prop.major, prop.minor);
		printf("Тактовая частота: %d\n", prop.clockRate);
		printf("Перекрытие копирования: ");
		if (prop.deviceOverlap)
			printf("Разрешено\n");
		else
			printf("Запрещено\n");
		printf("Тайм-аут выполнения ядра: ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Включен\n");
		else
			printf("Выключен\n");

		printf("--- Информация о памяти для устройства %d ---\n", i);
		printf("Всего глобальной памяти: %lld\n", prop.totalGlobalMem);
		printf("Всего константной памяти: %ld\n", prop.totalConstMem);
		printf("Максимальный шаг: %ld\n", prop.memPitch);
		printf("Выравнивание текстур: %ld\n", prop.textureAlignment);
		
		printf("--- Информация о мультипроцессорах для устройства %d ---\n", i);
		printf("Количество мультипроцессоров: %d\n", prop.multiProcessorCount);
		printf("Разделяемая память на один МП: %ld\n", prop.sharedMemPerBlock);
		printf("Регистров на один МП: %d\n", prop.regsPerBlock);
		printf("Нитей в варпе: %d\n", prop.warpSize);
		printf("Макс. количество нитей в блоке: %d\n", prop.maxThreadsPerBlock);
		printf("Макс. количество нитей по измерениям: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Максимальные размеры сетки: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		size_t free = 0;
		size_t total = 0;
		hipMemGetInfo (&free, &total);
		printf("Количество свободной памяти: %lld, всего %lld\n", free, total); 
		printf("\n");	
	}
	return 0;
}


// Main entry into the program 
int main(void) 
{ 
	setlocale (LC_ALL, ".1251"); // для вывода;

	int		i;
	int		option;			/* TEMPLATE LENGTH/STREAM LENGTH/GENERATOR*/
	char	*streamFile;	/* STREAM FILENAME     */

	DeviceInfo();

	// Определяем параметры теста (структура определена в defs.h)
	tp.n = 100; //60000000 int shared memory max. 
	tp.blockFrequencyBlockLength = 128;
	tp.nonOverlappingTemplateBlockLength = 9;
	tp.overlappingTemplateBlockLength = 9;
	tp.approximateEntropyBlockLength = 10;
	tp.serialBlockLength = 16;
	tp.linearComplexitySequenceLength = 500;
	tp.numOfBitStreams = 1;

	option = generatorOptions(&streamFile); // показываем генераторы
	chooseTests(); // выбираем тесты - все или по одному (?)
	fixParameters(); // пользователь может изменить дефолтные параметры
	openOutputStreams(option);
	invokeTestSuite(option, streamFile);

	printf("Press any key...\n");
	getchar();
	
/*	
	

	
	
	fclose(freqfp);
	for( i=1; i<=NUMOFTESTS; i++ ) {
		if ( stats[i] != NULL )
			fclose(stats[i]);
		if ( results[i] != NULL )
			fclose(results[i]);
	}
	if ( (testVector[0] == 1) || (testVector[TEST_CUSUM] == 1) ) 
		partitionResultFile(2, tp.numOfBitStreams, option, TEST_CUSUM);
	if ( (testVector[0] == 1) || (testVector[TEST_NONPERIODIC] == 1) ) 
		partitionResultFile(MAXNUMOFTEMPLATES, tp.numOfBitStreams, option, TEST_NONPERIODIC);
	if ( (testVector[0] == 1) || (testVector[TEST_RND_EXCURSION] == 1) )
		partitionResultFile(8, tp.numOfBitStreams, option, TEST_RND_EXCURSION);
	if ( (testVector[0] == 1) || (testVector[TEST_RND_EXCURSION_VAR] == 1) )
		partitionResultFile(18, tp.numOfBitStreams, option, TEST_RND_EXCURSION_VAR);
	if ( (testVector[0] == 1) || (testVector[TEST_SERIAL] == 1) )
		partitionResultFile(2, tp.numOfBitStreams, option, TEST_SERIAL);
	fprintf(summary, "------------------------------------------------------------------------------\n");
	fprintf(summary, "RESULTS FOR THE UNIFORMITY OF P-VALUES AND THE PROPORTION OF PASSING SEQUENCES\n");
	fprintf(summary, "------------------------------------------------------------------------------\n");
	fprintf(summary, "   generator is <%s>\n", streamFile);
	fprintf(summary, "------------------------------------------------------------------------------\n");
	fprintf(summary, " C1  C2  C3  C4  C5  C6  C7  C8  C9 C10  P-VALUE  PROPORTION  STATISTICAL TEST\n");
	fprintf(summary, "------------------------------------------------------------------------------\n");
	postProcessResults(option);
	fclose(summary);

	*/

	printf("Press any key...\n");
	getchar();
	return 0; 
} 
/*
void
partitionResultFile(int numOfFiles, int numOfSequences, int option, int testNameID)
{ 
	int		i, k, m, j, start, end, num, numread;
	float	c;
	FILE	**fp = (FILE **)calloc(numOfFiles+1, sizeof(FILE *));
	int		*results = (int *)calloc(numOfFiles, sizeof(int *));
	char	*s[MAXFILESPERMITTEDFORPARTITION];
	char	resultsDir[200];
	
	for ( i=0; i<MAXFILESPERMITTEDFORPARTITION; i++ )
		s[i] = (char*)calloc(200, sizeof(char));
	
	sprintf(resultsDir, "experiments/%s/%s/results.txt", generatorDir[option], testNames[testNameID]);
	
	if ( (fp[numOfFiles] = fopen(resultsDir, "r")) == NULL ) {
		printf("%s", resultsDir);
		printf(" -- file not found. Exiting program.\n");
		exit(-1);
	}
	
	for ( i=0; i<numOfFiles; i++ ) {
		if ( i < 10 )
			sprintf(s[i], "experiments/%s/%s/data%1d.txt", generatorDir[option], testNames[testNameID], i+1);
		else if (i < 100)
			sprintf(s[i], "experiments/%s/%s/data%2d.txt", generatorDir[option], testNames[testNameID], i+1);
		else
			sprintf(s[i], "experiments/%s/%s/data%3d.txt", generatorDir[option], testNames[testNameID], i+1);
	}
	numread = 0;
	m = numOfFiles/20;
	if ( (numOfFiles%20) != 0 )
		m++;
	for ( i=0; i<numOfFiles; i++ ) {
		if ( (fp[i] = fopen(s[i], "w")) == NULL ) {
			printf("%s", s[i]);
			printf(" -- file not found. Exiting program.\n");
			exit(-1);
		}
		fclose(fp[i]);
	}
	for ( num=0; num<numOfSequences; num++ ) {
		for ( k=0; k<m; k++ ) { 			// FOR EACH BATCH 
			
			start = k*20;		// BOUNDARY SEGMENTS
			end   = k*20+19;
			if ( k == (m-1) )
				end = numOfFiles-1;
			
			for ( i=start; i<=end; i++ ) {		// OPEN FILE 
				if ( (fp[i] = fopen(s[i], "a")) == NULL ) {
					printf("%s", s[i]);
					printf(" -- file not found. Exiting program.\n");
					exit(-1);
				}
			}
			
			for ( j=start; j<=end; j++ ) {		// POPULATE FILE 
				fscanf(fp[numOfFiles], "%f", &c);
				fprintf(fp[j], "%f\n", c);
				numread++;
			}

			for ( i=start; i<=end; i++ )		// CLOSE FILE
				fclose(fp[i]);
		}
	}
	fclose(fp[numOfFiles]);
	for ( i=0; i<MAXFILESPERMITTEDFORPARTITION; i++ )
		free(s[i]);

	return;
}

int
cmp(const double *a, const double *b)
{
	if ( *a < *b )
		return -1;
	if ( *a == *b )
		return 0;
	return 1;
}


void
postProcessResults(int option)
{
	int		i, k, randomExcursionSampleSize, generalSampleSize;
	int		passRate, case1, case2, numOfFiles = 2;
	double	p_hat;
	char	s[200];
	
	for ( i=1; i<=NUMOFTESTS; i++ ) {		// FOR EACH TEST
		if ( testVector[i] ) {
			// SPECIAL CASES -- HANDLING MULTIPLE FILES FOR A SINGLE TEST
			if ( ((i == TEST_CUSUM) && testVector[TEST_CUSUM] ) ||
				 ((i == TEST_NONPERIODIC) && testVector[TEST_NONPERIODIC] ) ||
				 ((i == TEST_RND_EXCURSION) && testVector[TEST_RND_EXCURSION]) ||
				 ((i == TEST_RND_EXCURSION_VAR) && testVector[TEST_RND_EXCURSION_VAR]) || 
				 ((i == TEST_SERIAL) && testVector[TEST_SERIAL]) ) {
				
				if ( (i == TEST_NONPERIODIC) && testVector[TEST_NONPERIODIC] )  
					numOfFiles = MAXNUMOFTEMPLATES;
				else if ( (i == TEST_RND_EXCURSION) && testVector[TEST_RND_EXCURSION] ) 
					numOfFiles = 8;
				else if ( (i == TEST_RND_EXCURSION_VAR) && testVector[TEST_RND_EXCURSION_VAR] ) 
					numOfFiles = 18;
				else
					numOfFiles = 2;
				for ( k=0; k<numOfFiles; k++ ) {
					if ( k < 10 )
						sprintf(s, "experiments/%s/%s/data%1d.txt", generatorDir[option], testNames[i], k+1);
					else if ( k < 100 )
						sprintf(s, "experiments/%s/%s/data%2d.txt", generatorDir[option], testNames[i], k+1);
					else
						sprintf(s, "experiments/%s/%s/data%3d.txt", generatorDir[option], testNames[i], k+1);
					if ( (i == TEST_RND_EXCURSION) || (i == TEST_RND_EXCURSION_VAR) ) 
						randomExcursionSampleSize = computeMetrics(s,i);
					else
						generalSampleSize = computeMetrics(s,i);
				}
			}
			else {
				sprintf(s, "experiments/%s/%s/results.txt", generatorDir[option], testNames[i]);
				generalSampleSize = computeMetrics(s,i);
			}
		}
	}

	fprintf(summary, "\n\n- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
	case1 = 0;
	case2 = 0;
	if ( testVector[TEST_RND_EXCURSION] || testVector[TEST_RND_EXCURSION_VAR] ) 
		case2 = 1;
	for ( i=1; i<=NUMOFTESTS; i++ ) {
		if ( testVector[i] && (i != TEST_RND_EXCURSION) && (i != TEST_RND_EXCURSION_VAR) ) {
			case1 = 1;
			break;
		}
	}
	p_hat = 1.0 - ALPHA;
	if ( case1 ) {
		if ( generalSampleSize == 0 ) {
			fprintf(summary, "The minimum pass rate for each statistical test with the exception of the\n");
			fprintf(summary, "random excursion (variant) test is undefined.\n\n");
		}
		else {
			passRate = (p_hat - 3.0 * sqrt((p_hat*ALPHA)/generalSampleSize)) * generalSampleSize;
			fprintf(summary, "The minimum pass rate for each statistical test with the exception of the\n");
			fprintf(summary, "random excursion (variant) test is approximately = %d for a\n", generalSampleSize ? passRate : 0);
			fprintf(summary, "sample size = %d binary sequences.\n\n", generalSampleSize);
		}
	}
	if ( case2 ) {
		if ( randomExcursionSampleSize == 0 )
			fprintf(summary, "The minimum pass rate for the random excursion (variant) test is undefined.\n\n");
		else {
			passRate = (p_hat - 3.0 * sqrt((p_hat*ALPHA)/randomExcursionSampleSize)) * randomExcursionSampleSize;
			fprintf(summary, "The minimum pass rate for the random excursion (variant) test\n");
			fprintf(summary, "is approximately = %d for a sample size = %d binary sequences.\n\n", passRate, randomExcursionSampleSize);
		}
	}
	fprintf(summary, "For further guidelines construct a probability table using the MAPLE program\n");
	fprintf(summary, "provided in the addendum section of the documentation.\n");
	fprintf(summary, "- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
}

int
computeMetrics(char *s, int test)
{
	int		j, pos, count, passCount, sampleSize, expCount, proportion_threshold_min, proportion_threshold_max;
	int		freqPerBin[10] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	double	*A, *T, chi2, proportion, uniformity, p_hat, tmp;
	float	c;
	FILE	*fp;
	
	if ( (fp = fopen(s, "r")) == NULL ) {
		printf("%s",s);
		printf(" -- file not found. Exiting program.\n");
		exit(-1);
	}
	
	if ( (A = (double *)calloc(tp.numOfBitStreams, sizeof(double))) == NULL ) {
		printf("Final Analysis Report aborted due to insufficient workspace\n");
		return 0;
	}
	
	// Compute Metric 1: Proportion of Passing Sequences 
	
	count = 0; 		
	sampleSize = tp.numOfBitStreams;
	
	if ( (test == TEST_RND_EXCURSION) || (test == TEST_RND_EXCURSION_VAR) ) { // Special Case: Random Excursion Tests 
		if ( (T = (double *)calloc(tp.numOfBitStreams, sizeof(double))) == NULL ) {
			printf("Final Analysis Report aborted due to insufficient workspace\n");
			return 0;
		}
		for ( j=0; j<sampleSize; j++ ) {
			fscanf(fp, "%f", &c);
			if ( c > 0.000000 )
				T[count++] = c;
		}
		
		if ( (A = (double *)calloc(count, sizeof(double))) == NULL ) {
			printf("Final Analysis Report aborted due to insufficient workspace\n");
			return 0;
		}
		
		for ( j=0; j<count; j++ )
			A[j] = T[j];
		
		sampleSize = count;
		count = 0;
		for ( j=0; j<sampleSize; j++ )
			if ( A[j] < ALPHA )
				count++;
		free(T);
	}
	else {
		if ( (A = (double *)calloc(sampleSize, sizeof(double))) == NULL ) {
			printf("Final Analysis Report aborted due to insufficient workspace\n");
			return 0;
		}
		for ( j=0; j<sampleSize; j++ ) {
			fscanf(fp, "%f", &c);
			if ( c < ALPHA )
				count++;
			A[j] = c;
		}
	}
	if ( sampleSize == 0 )
		passCount = 0;
	else
		passCount = sampleSize - count;
	
	p_hat = 1.0 - ALPHA;
	proportion_threshold_max = (p_hat + 3.0 * sqrt((p_hat*ALPHA)/sampleSize)) * sampleSize;
	proportion_threshold_min = (p_hat - 3.0 * sqrt((p_hat*ALPHA)/sampleSize)) * sampleSize;
	
	// Compute Metric 2: Histogram 
	
	//qsort((void *)A, sampleSize, sizeof(double), (void *)cmp); --------------------------------------------------------------------------------- ПОЧИНИТЬ!

	for ( j=0; j<sampleSize; j++ ) {
		pos = (int)floor(A[j]*10);
		if ( pos == 10 )
			pos--;
		freqPerBin[pos]++;
	}
	chi2 = 0.0;
	expCount = sampleSize/10;
	if ( expCount == 0 )
		uniformity = 0.0;
	else {
		for ( j=0; j<10; j++ )
			chi2 += pow(freqPerBin[j]-expCount, 2)/expCount;
		uniformity = cephes_igamc(9.0/2.0, chi2/2.0);
	}
	
	for ( j=0; j<10; j++ )			// DISPLAY RESULTS 
		fprintf(summary, "%3d ", freqPerBin[j]);
	
	if ( expCount == 0 )
		fprintf(summary, "    ----    ");
	else if ( uniformity < 0.0001 )
		fprintf(summary, " %8.6f * ", uniformity);
	else
		fprintf(summary, " %8.6f   ", uniformity);
	
	if ( sampleSize == 0 )
		fprintf(summary, " ------     %s\n", testNames[test]);
	//	else if ( proportion < 0.96 )
	else if ( (passCount < proportion_threshold_min) || (passCount > proportion_threshold_max))
		fprintf(summary, "%4d/%-4d *  %s\n", passCount, sampleSize, testNames[test]);
	else
		fprintf(summary, "%4d/%-4d    %s\n", passCount, sampleSize, testNames[test]);
	
	fclose(fp);
	free(A);
	
	return sampleSize;
}
*/